/*
 * Really, really primitive DNA sequence aligner meant to demonstrate the
 * kinds of answers we want from your aligners.  This uses a brute-force
 * algorithm that can be greatly improved.
 *
 * Stephen A. Edwards, 2012
 *
 * Sample usage:
 *
 * ./align human_g1k_v37.bin ERR050082.filt.bin 100 0 9999
 *
 * The first argument is the reference genome in packed binary form.
 *
 * The second argument is the sequence file, also in packed binary form.
 *
 * The third argument is the number of base pairs per sequence
 *
 * The fourth (optional) argument is the starting sequence number
 *
 * The fifth (optional) argument is the ending sequence number
 *
 * If the ending sequence number is omitted, it is taken as the end of
 * the file.
 *
 * If the starting sequence number is omitted, it is taken as the
 * beginning of the file.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <unistd.h>
#include <string.h>

/* Assumed alignment for sequences in the sequence file, in bytes */
#define SEQUENCE_ALIGN 8

/* Result of matching a single sequence; match() fills in an array */
struct match {
  long int position;      /* Position of a match if count > 0 */ //     off_t is an internal variable that limits the max length you can read from a file as an  
                                                                 //     offset from the current file pointer 
  unsigned int count;  /* Number of matches found */
};

/* For debugging: print a packed sequence */
void dump_sequence(char *bases, int bytes)
{
  static char base[] = {'A', 'C', 'G', 'T'};
  int i;
  for (i = 0 ; i != bytes; i++) {
      char c = bases[i];                                    // is this similar to doing c = *(bases+i)
      printf("%c%c%c%c",
	     base[c&0x3],
	     base[(c >> 2) & 0x3],
	     base[(c >> 4) & 0x3],
	     base[(c >> 6) & 0x3]);
    }
    printf("\n");
}

/* Attempt to match an array of two-bit sequences (*sequences)
 * against a reference sequence (*reference_base).
 *
 * Fill in the *matches array with the results.
 *
 * This allocates a buffer large enough to hold a single sequence
 * and shifts the whole reference sequence through it, one pair at a time.
 * At each point, this buffer is compared to each of the given sequences.
 */

                                  
__global__ void matchKernel(char *ref_genome_GPU, char *read_seq_GPU, struct match *result_GPU, long int reference_total,
int sequence_length,int N){

 char ref[25];

 long int p,last_p;

 __shared__ char seq[25];
 int i,j,count=3;

 for(i=0;i<N;i++)
 result_GPU[blockIdx.x].count = 0;

 /*Copy first 100 base pairs from global memory to block shared memory*/
  for(i=0;i<25;i++)
  ref[i]=ref_genome_GPU[i];
 
 
 /*Copy read sequence from global memory to internal register */

  for(i=0;i<25;i++)
  seq[i]= read_seq_GPU[blockIdx.x*32 + i];
  
 last_p = (reference_total*4) - sequence_length;

 for(p=0;p<last_p;p++)
   {
    char temp=0,read_match=1,shift_in=0,shift_out=0;
 
    if(count==-1)count=3;
    
      for(i=0;i<25;i++){
         
         if(seq[i]!=ref[i]){
         read_match=0;
         break;
         }            
       }
    if(read_match){
             result_GPU[blockIdx.x].count++;
             result_GPU[blockIdx.x].position=p;    
             }
 /*To get new base pair at the last 2 bits of ref[], swap first two BP with last 2 BP of previous byte moveing from end of  
   reference array to the start*/

   for(i=24;i>-1;i--){
       shift_out=(ref[i]<<6)&0xC0;
       ref[i] = ((ref[i]>>2)&0x3F)| shift_in;
       shift_in = shift_out;
      }
  
  

 /*Attach two BP from the next byte of the reference genome to the last 2 bits of the last byte of reference buffer*/   
   
   ref[24]&=0x3F;
   temp = ref_genome_GPU[(p+100)>>2]<<(2*count);
   ref[24]|=temp&0xC0;
   count--;
  }

}

void match(struct match *matches, /* Output array, one per sequence */
	   char *reference_base,  /* Beginning of reference sequence */
	   long int reference_total, /* Number of bytes in the reference */
	   char *sequences,       /* Beginning of sequences */
	   int start_sequence,    /* Index of first sequence in array */
	   int sequence_length,   /* Base pairs per sequence */
	   int end_sequence)      /* One more than index of last sequence */
{
  /* Bytes per sequence, padded to be a multiple of SEQUENCE_ALIGN */
  int sequence_bytes = ((sequence_length + SEQUENCE_ALIGN * 4 - 1) & 
                        ~(SEQUENCE_ALIGN*4 - 1)) >> 2;
  int N = end_sequence-start_sequence;
 
  /*Declare pointers to the GPU memory*/

   char *ref_genome_GPU;
   char *read_seq_GPU;
   struct match *result_GPU;

   dim3 dimBlock(1,1,1);
   dim3 dimGrid(N,1,1);
   

  /*Allocate memory on GPU */

  hipMalloc((void**) &ref_genome_GPU, reference_total);
  hipMalloc((void**) &read_seq_GPU, N*sequence_bytes);
  hipMalloc((void**) &result_GPU, sizeof(struct match)*N);

  /*copy the Reference Genome and read sequences from host to GPU global memory*/

  hipMemcpy(ref_genome_GPU,reference_base,reference_total,hipMemcpyHostToDevice);
  hipMemcpy(read_seq_GPU,sequences,N*sequence_bytes,hipMemcpyHostToDevice);

  
  /*Kernel call*/

 matchKernel<<<dimGrid,dimBlock>>>(ref_genome_GPU,read_seq_GPU, result_GPU, reference_total, sequence_length,N);


  /*Copy result from GPU memory to host*/
  
 hipMemcpy(matches,result_GPU, sizeof(struct match)*N,hipMemcpyDeviceToHost);

 /*Free the GPU memory*/
  
 hipFree(ref_genome_GPU);
 hipFree(read_seq_GPU);
 hipFree(result_GPU);
 
 

}

/*
 * Process command-line arguments, map reference and sequence data into
 * memory, call match(), and print the results
 */

int main(int argc, const char *argv[])
{
  const char *reference_filename, *sequence_filename;                           // declare pointers to reference and read files
 

  int reference_fd = -1, sequence_fd = -1;                                     // reference_fd & sequence_fd are used as a file handles to reference & read files
   
 /*sequence length used for storing the length of one read
   sequence bytes is the padded length of one read
   start and end sequences store the start and end index to do the matching
   num sequecne denotess the number of sequences in one read file*/

 int sequence_length, sequence_bytes, start_sequence = -1, end_sequence = -1,
     num_sequences, i;

  
  long int reference_total, sequence_total, sequence_window_offset;
  
  size_t sequence_window_length;                                                // size_t is a internal type that any defined type is later mapped to for use in 
                                                                                //functions like strlen(), malloc and so on
  long int page_size;

  void *reference_base, *sequence_base;

  char *sequences;
  
  struct stat file_status;                                          // The stat structure type is used to return information about the attributes of a file. 

  struct match *matches;
  

  /*On modern operating systems, it is possible to mmap (pronounced "em-map") a file to a region of memory. When this is done, the file can be accessed just like an array in the program. Since mmapped pages can be stored back to their file when physical memory is low, it is possible to mmap files orders of magnitude larger than both the physical memory and swap space. The only limit is address space. Memory mapping only works on entire pages of memory. Thus, addresses for mapping     must be page-aligned, and length values will be rounded up. To determine the size of a page the machine uses one should use
size_t page_size = (size_t) sysconf (_SC_PAGESIZE);*/

  page_size = sysconf(_SC_PAGE_SIZE);                              /* needed for mmap */

  if (argc < 4) goto usage;

  reference_filename = argv[1];                                    // point reference file pointer to the first argument
  sequence_filename = argv[2];                                     // point read file pointer to the second argument

  sequence_length = atoi(argv[3]);                                 // sequence length stores the length of one read

  /* Pad the sequences out to a multiple of SEQUENCE_ALIGN bytes */
  sequence_bytes = ((sequence_length + SEQUENCE_ALIGN * 4 - 1) & 
		    ~(SEQUENCE_ALIGN * 4 - 1)) >> 2;               // sequence length = 100, sequence bytes is 32

  if (sequence_length <= 0) {
    fprintf(stderr,
	    "Error: given sequence length must be an integer greater than zero\n");
    goto usage;
  }

  if ((reference_fd = open(reference_filename, O_RDONLY)) < 0) {      // check if reference genome file opened correctly
    fprintf(stderr, "Error opening reference file \"%s\": ",
	    reference_filename);
    perror((const char *) 0);
    goto usage;
  }

  if (fstat(reference_fd, &file_status)) {                           // check if the reference genome file status is correct.
    fprintf(stderr, "Error checking reference file \"%s\": ",
	    reference_filename);
    perror((const char *) 0);
    goto usage;
  }

  reference_total = file_status.st_size;                                   // file_status.st_size returns the size of the reference genome file

  if (reference_total < sequence_bytes) {                                  // check if the genome is bigger than the read
    fprintf(stderr, "Error: reference file is shorter than the given sequence length (%d)\n", sequence_length);
    goto usage;
  }

 
  if ((sequence_fd = open(sequence_filename, O_RDONLY)) < 0) {             // check if read sequence file opened correctly
    fprintf(stderr, "Error opening sequence file \"%s\": ", sequence_filename);
    perror((const char *) 0);
    goto usage;
  }

  if (fstat(sequence_fd, &file_status)) {                                 // check if read sequence file status is correct
    fprintf(stderr, "Error checking sequence file \"%s\": ", sequence_filename);
    perror((const char *) 0);
    goto usage;
  }
 
  sequence_total = file_status.st_size;                                    // assign size of sequence file to sequence total 

  if (sequence_total < sequence_bytes) {                                   // check if the read sequence if biggeer than user input
    fprintf(stderr, "Sequence file is too small\n");
    goto usage;
  }

  if (sequence_total % sequence_bytes != 0)                                
    fprintf(stderr, "Warning: sequence file may be truncated\n");

  num_sequences = sequence_total / sequence_bytes;                          // calculate total number of read sequences

  if (argc > 4) start_sequence = atoi(argv[4]);
  if (start_sequence < 0) start_sequence = 0;
  if (start_sequence >= num_sequences) {
    fprintf(stderr, "Error: initial sequence number must be less than %d\n",
	    num_sequences);
    goto usage;
  }

  if (argc > 5) end_sequence = atoi(argv[5]);
  else end_sequence = num_sequences;
  if (end_sequence < start_sequence || end_sequence > num_sequences) {
    fprintf(stderr, "Error: End sequence number must be between %d and %d\n",
	    start_sequence, num_sequences);
    goto closeexit;
  }

  /* mmap the reference data */
  /*pa=mmap(addr, len, prot, flags, fildes, off);

  The mmap() function shall establish a mapping between the address space of the process at an address pa for len bytes to the memory object represented by the file 
  descriptor fildes at offset off for len bytes. */
  reference_base = mmap( (void*) 0, reference_total, PROT_READ, MAP_SHARED,
			 reference_fd, 0);

  if (reference_base == MAP_FAILED) {
    perror("Error when attempting to map the reference file");
    goto closeexit;
  }

  /* mmap the sequence data */

  /* compute the starting location by rounding down to the nearest
     page boundary; window length is the difference between this and the last
     page on which the sequences fall */
  sequence_window_offset = (start_sequence * sequence_bytes) & ~(page_size - 1);
  sequence_window_length = (((end_sequence * sequence_bytes) + (page_size - 1))
			    & ~(page_size - 1)) - sequence_window_offset;

  sequence_base = mmap( (void *) 0, sequence_window_length, PROT_READ,
			MAP_SHARED, sequence_fd, sequence_window_offset);

  
 

  if (sequence_base == MAP_FAILED) {
    perror("Error when attempting to map the sequence file");
    goto closeexit;
  }
 sequences = (char*)sequence_base + ((start_sequence * sequence_bytes) - sequence_window_offset);

  /* Allocate space to hold the results of matching */

  matches = (struct match*)malloc(sizeof(struct match)*(end_sequence - start_sequence));
  if (matches == NULL) {
    fprintf(stderr, "Failed to allocate memory for match information\n");
    goto unmap_sequences;
  }

  for ( i = 0 ; i < end_sequence - start_sequence ; i++ )
    matches[i].count = 0;

  
  match(matches,(char *)reference_base, reference_total, (char *)sequences, start_sequence, sequence_length, end_sequence);

  /* Report all matches */
  for (i = start_sequence ; i != end_sequence ; ++i) {
    printf("%8d: ", i);
    if (matches[i - start_sequence].count) {
      printf("%9ld", matches[i - start_sequence].position);
      if (matches[i-start_sequence].count > 1)
	printf(" + %d others", matches[i-start_sequence].count - 1);
      printf("\n");
    } else
      printf("-\n");
  }

  free(matches);

 unmap_sequences:
  if (munmap(sequence_base, sequence_window_length)) {
    perror("Error when unmapping the sequence file");
    goto closeexit;
  }

 unmap_references:
  if (munmap(reference_base, reference_total)) {
    perror("Error when unmapping the reference file");
    goto closeexit;
  }

  close(sequence_fd);
  close(reference_fd);
  return 0;

 usage:
  fprintf(stderr,
	  "usage: align <reference-genome> <sequence-file> <sequence-length> <start> <end>\n"
	  "<reference-genome> is the name of a packed binary reference sequence.\n"
	  "<sequence-file> is the name of a packed binary sequence file.\n"
	  "<sequence-length> is an integer indicating the length, in base pairs, of each sequence.\n"
	  "<start> is the optional starting sequence number.  If omitted, it defaults to the start of the sequence file.\n"
	  "<end> is the optional ending sequence number.  If omitted, it defaults to the end of the sequence file.\n");
 closeexit:
  if (reference_fd >= 0) close(reference_fd);
  if (sequence_fd >= 0) close(sequence_fd);
  return 1;
}
